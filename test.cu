#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include "cpu_encoder.h"
#include "gpu_encoder_v1.h"
#include "gpu_encoder_v2.h"
#include "gpu_encoder_v3.h"


using namespace std::chrono;

template <typename T, typename T_fp, typename T_sfp, typename T_bp>
void test_cpu(int n, int b) {


    printf("Begin test\n");
    T * data = new T[n];
    T * decoded_data = new T[n];
    int bitplane_length = n/(sizeof(T_bp)*8)*2;
    T_bp * encoded_bitplanes = new T_bp[bitplane_length*b];
    T max_abs = 0;
    int exp = 0;
    for (int i = 0; i < n; i++) {
        data[i] = i;
        if (fabs(data[i] > max_abs)) {
            max_abs = fabs(data[i]);
        }
        
    }
    frexp(max_abs, &exp);
    
    printf("Encode: \n");
    auto start = std::chrono::high_resolution_clock::now();
    cpu::encode<T, T_fp, T_sfp, T_bp>(data, encoded_bitplanes, n, b, exp);
    auto stop = std::chrono::high_resolution_clock::now();
    auto time = std::chrono::duration_cast<microseconds>(stop - start).count();
    printf("time: %f s, %f GB/s\n", float(time)/1e6, float(n * sizeof(T))/1e3/time);


    printf("Decode: \n");
    start = std::chrono::high_resolution_clock::now();
    cpu::decode<T, T_fp, T_sfp, T_bp>(decoded_data, encoded_bitplanes, n, b, exp);
    stop = std::chrono::high_resolution_clock::now();
    time = std::chrono::duration_cast<microseconds>(stop - start).count();
    printf("time: %f s, %f GB/s\n", float(time)/1e6, float(n * sizeof(T))/1e3/time);

    bool pass = true;
    for (int i = 0; i < n; i++) {
        if (data[i] !=  decoded_data[i]) {
            pass = false;
            // printf("%f, %f\n", data[i], decoded_data[i]);
            break;
        }
    }
    
    delete[] data;
    delete[] decoded_data;
    delete[] encoded_bitplanes;

    printf("Pass: %d\n", pass);

    printf("Finish test\n");
}
template <typename T, typename T_fp, typename T_sfp, typename T_bp>
void test_gpu_v1(int n, int b) {
    printf("Begin test\n");
    T * data = new T[n];
    T * decoded_data = new T[n];
    int bitplane_length = n/(sizeof(T_bp)*8)*2;
    T_bp * encoded_bitplanes = new T_bp[bitplane_length*b];

    T * d_data;
    T * d_decoded_data;
    T_bp * d_encoded_bitplanes;
    hipMalloc(&d_data, n*sizeof(T));
    hipMalloc(&d_decoded_data, n*sizeof(T));
    hipMalloc(&d_encoded_bitplanes, bitplane_length*b*sizeof(T_bp));

    T max_abs = 0;
    int exp = 0;
    for (int i = 0; i < n; i++) {
        data[i] = i;
        if (fabs(data[i]) > max_abs) {
            max_abs = fabs(data[i]);
        }
        
    }
    frexp(max_abs, &exp);

    hipMemcpy(d_data, data, n*sizeof(T), hipMemcpyDefault);

    // warmup gpu
    for (int i = 0; i < 10; i++) {
        gpu_v1::encode<T, T_fp, T_sfp, T_bp> <<<(n-1)/256+1, 256>>>(d_data, d_encoded_bitplanes, n, b, exp);
    }

    printf("Encode: \n");
    hipDeviceSynchronize();
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < 10; i++) {
        gpu_v1::encode<T, T_fp, T_sfp, T_bp> <<<(n-1)/256+1, 256>>>(d_data, d_encoded_bitplanes, n, b, exp);
    }
    hipDeviceSynchronize();
    auto stop = std::chrono::high_resolution_clock::now();
    auto time = std::chrono::duration_cast<microseconds>(stop - start).count() / 10;
    printf("time: %f s, %f GB/s\n", float(time)/1e6, float(n * sizeof(T))/1e3/time);

    hipMemcpy(encoded_bitplanes, d_encoded_bitplanes, bitplane_length*b*sizeof(T_bp), hipMemcpyDefault);
    hipMemcpy(d_encoded_bitplanes, encoded_bitplanes, bitplane_length*b*sizeof(T_bp), hipMemcpyDefault);

    printf("Decode: \n");
    hipDeviceSynchronize();
    start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < 10; i++) {
        gpu_v1::decode<T, T_fp, T_sfp, T_bp> <<<(n-1)/256+1, 256>>>(d_decoded_data, d_encoded_bitplanes, n, b, exp);
    }
    hipDeviceSynchronize();
    stop = std::chrono::high_resolution_clock::now();
    time = std::chrono::duration_cast<microseconds>(stop - start).count() / 10;
    printf("time: %f s, %f GB/s\n", float(time)/1e6, float(n * sizeof(T))/1e3/time);

    hipMemcpy(decoded_data, d_decoded_data, n*sizeof(T), hipMemcpyDefault);

    bool pass = true;
    for (int i = 0; i < n; i++) {
        // printf("%f, %f\n", data[i], decoded_data[i]);
        if (data[i] !=  decoded_data[i]) {
            pass = false;
            break;
        }
    }

    delete[] data;
    delete[] decoded_data;
    delete[] encoded_bitplanes;

    hipFree(d_data);
    hipFree(d_decoded_data);
    hipFree(d_encoded_bitplanes);

    printf("Pass: %d\n", pass);

    printf("Finish test\n");
}

template <typename T, typename T_fp, typename T_sfp, typename T_bp>
void test_gpu_v2(int n, int b) {
    printf("Begin test\n");
    T *data = new T[n];
    T *decoded_data = new T[n];
    constexpr int batch_size = sizeof(T_bp) * 8;
    const int num_batches = (n + batch_size - 1) / batch_size;
    const int bitplane_length = num_batches * 2 * b; // Correct memory space allocation
    T_bp *encoded_bitplanes = new T_bp[bitplane_length];

    T *d_data;
    T *d_decoded_data;
    T_bp *d_encoded_bitplanes;
    hipMalloc(&d_data, n * sizeof(T));
    hipMalloc(&d_decoded_data, n * sizeof(T));
    hipMalloc(&d_encoded_bitplanes, bitplane_length * sizeof(T_bp));

    T max_abs = 0;
    int exp = 0;
    for (int i = 0; i < n; i++) {
        data[i] = i;
        if (fabs(data[i]) > max_abs) {
            max_abs = fabs(data[i]);
        }
    }
    frexp(max_abs, &exp);

    hipMemcpy(d_data, data, n * sizeof(T), hipMemcpyDefault);

    // Warmup GPU
    for (int i = 0; i < 10; i++) {
        gpu_v2::encode<T, T_fp, T_sfp, T_bp><<<num_batches, batch_size>>>(d_data, d_encoded_bitplanes, n, b, exp);
    }

    printf("Encode (GPU V2): \n");
    hipDeviceSynchronize();
    auto start = high_resolution_clock::now();
    for (int i = 0; i < 10; i++) {
        gpu_v2::encode<T, T_fp, T_sfp, T_bp><<<num_batches, batch_size>>>(d_data, d_encoded_bitplanes, n, b, exp);
    }
    hipDeviceSynchronize();
    auto stop = high_resolution_clock::now();
    auto time = duration_cast<microseconds>(stop - start).count() / 10;
    printf("time: %f s, %f GB/s\n", float(time) / 1e6, float(n * sizeof(T)) / 1e3 / time);

    hipMemcpy(encoded_bitplanes, d_encoded_bitplanes, bitplane_length * sizeof(T_bp), hipMemcpyDefault);

    printf("Decode (GPU V2): \n");
    hipDeviceSynchronize();
    start = high_resolution_clock::now();
    for (int i = 0; i < 10; i++) {
        gpu_v2::decode<T, T_fp, T_sfp, T_bp><<<num_batches, batch_size>>>(d_decoded_data, d_encoded_bitplanes, n, b, exp);
    }
    hipDeviceSynchronize();
    stop = high_resolution_clock::now();
    time = duration_cast<microseconds>(stop - start).count() / 10;
    printf("time: %f s, %f GB/s\n", float(time) / 1e6, float(n * sizeof(T)) / 1e3 / time);

    hipMemcpy(decoded_data, d_decoded_data, n * sizeof(T), hipMemcpyDefault);

    bool pass = true;
    for (int i = 0; i < n; i++) {
        if (data[i] != decoded_data[i]) {
            pass = false;
            printf("Mismatch at %d: original=%f, decoded=%f\n", i, data[i], decoded_data[i]);
            break;
        }
    }

    delete[] data;
    delete[] decoded_data;
    delete[] encoded_bitplanes;
    hipFree(d_data);
    hipFree(d_decoded_data);
    hipFree(d_encoded_bitplanes);

    printf("Pass: %d\n", pass);
    printf("Finish test\n");
}


template <typename T, typename T_fp, typename T_sfp, typename T_bp>
void test_gpu_v3(int n, int b) {
    printf("Begin test\n");
    T *data = new T[n];
    T *decoded_data = new T[n];
    constexpr int batch_size = sizeof(T_bp) * 8;
    const int num_batches = (n + batch_size - 1) / batch_size;
    const int bitplane_length = num_batches * 2 * b; // Correct memory space allocation
    T_bp *encoded_bitplanes = new T_bp[bitplane_length];

    T *d_data;
    T *d_decoded_data;
    T_bp *d_encoded_bitplanes;
    hipMalloc(&d_data, n * sizeof(T));
    hipMalloc(&d_decoded_data, n * sizeof(T));
    hipMalloc(&d_encoded_bitplanes, bitplane_length * sizeof(T_bp));

    T max_abs = 0;
    int exp = 0;
    for (int i = 0; i < n; i++) {
        data[i] = i;
        if (fabs(data[i]) > max_abs) {
            max_abs = fabs(data[i]);
        }
    }
    frexp(max_abs, &exp);

    hipMemcpy(d_data, data, n * sizeof(T), hipMemcpyDefault);

    const int warps_per_block = 4;
    const int batches_per_warp = 2;
    const int block_size = warps_per_block * 32;

    const int total_warps = (num_batches + batches_per_warp - 1) / batches_per_warp;
    const int num_blocks = (total_warps + warps_per_block - 1) / warps_per_block;

    // Warmup GPU
    for (int i = 0; i < 10; i++) {
        gpu_v3::encode<T, T_fp, T_sfp, T_bp><<<num_blocks, block_size>>>(d_data, d_encoded_bitplanes, batches_per_warp, n, b, exp);
    }

    printf("Encode (GPU V3): \n");
    hipDeviceSynchronize();
    auto start = high_resolution_clock::now();
    for (int i = 0; i < 10; i++) {
        gpu_v3::encode<T, T_fp, T_sfp, T_bp><<<num_blocks, block_size>>>(d_data, d_encoded_bitplanes, batches_per_warp, n, b, exp);
    }
    hipDeviceSynchronize();
    auto stop = high_resolution_clock::now();
    auto time = duration_cast<microseconds>(stop - start).count() / 10;
    printf("time: %f s, %f GB/s\n", float(time) / 1e6, float(n * sizeof(T)) / 1e3 / time);

    hipMemcpy(encoded_bitplanes, d_encoded_bitplanes, bitplane_length * sizeof(T_bp), hipMemcpyDefault);

    printf("Decode (GPU V3): \n");
    hipDeviceSynchronize();
    start = high_resolution_clock::now();
    for (int i = 0; i < 10; i++) {
        gpu_v3::decode<T, T_fp, T_sfp, T_bp><<<num_blocks, block_size>>>(d_decoded_data, d_encoded_bitplanes, batches_per_warp, n, b, exp);
    }
    hipDeviceSynchronize();
    stop = high_resolution_clock::now();
    time = duration_cast<microseconds>(stop - start).count() / 10;
    printf("time: %f s, %f GB/s\n", float(time) / 1e6, float(n * sizeof(T)) / 1e3 / time);

    hipMemcpy(decoded_data, d_decoded_data, n * sizeof(T), hipMemcpyDefault);

    bool pass = true;
    for (int i = 0; i < n; i++) {
        if (data[i] != decoded_data[i]) {
            pass = false;
            printf("Mismatch at %d: original=%f, decoded=%f\n", i, data[i], decoded_data[i]);
            break;
        }
    }

    delete[] data;
    delete[] decoded_data;
    delete[] encoded_bitplanes;
    hipFree(d_data);
    hipFree(d_decoded_data);
    hipFree(d_encoded_bitplanes);

    printf("Pass: %d\n", pass);
    printf("Finish test\n");
}


template <typename T>
void test(int n, int b) {
    using T_sfp = typename std::conditional<std::is_same<T, double>::value,
                                          int64_t, int32_t>::type;
    using T_fp = typename std::conditional<std::is_same<T, double>::value,
                                            uint64_t, uint32_t>::type;
    using T_bp = uint32_t;


    test_cpu<float, T_fp, T_sfp, T_bp>(n, b);
    test_gpu_v1<float, T_fp, T_sfp, T_bp>(n, b);
    test_gpu_v2<float, T_fp, T_sfp, T_bp>(n, b);
    test_gpu_v3<float, T_fp, T_sfp, T_bp>(n, b);
}


int main() {
    int b = 32;
    
    // n needs to be a multiple of 32
    int n15 = 128 * 1024;
    int n17 = 512 * 1024;
    int n21 = 2 * 1024 * 1024;
    int n23 = 8 * 1024 * 1024;
    int n25 = 32 * 1024 * 1024;
    test<float>(n15, b);
    test<float>(n17, b);
    test<float>(n21, b);
    test<float>(n23, b);
    test<float>(n25, b); 
}